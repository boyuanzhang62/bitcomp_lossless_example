/*
 * Copyright (c) 2023, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// Simple example to show how to use bitcomp's native lossy API to compress
// floating point data.
//
// Bitcomp's lossy compression performs an on-the-fly integer quantization
// and compresses the resulting integral values with the lossless encoder.
// A smaller delta used for the quantization will typically lower the
// compression ratio, but will increase precision.

#include <algorithm>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string>

#include <native/bitcomp.h>

#include "utils.h"

#define CUDA_CHECK(func)                                                        \
    do                                                                          \
    {                                                                           \
        hipError_t rt = (func);                                                \
        if (rt != hipSuccess)                                                  \
        {                                                                       \
            std::cout << "API call failure \"" #func "\" with " << rt << " at " \
                      << __FILE__ << ":" << __LINE__ << std::endl;              \
            throw;                                                              \
        }                                                                       \
    } while (0);

#define BITCOMP_CHECK(call)                                    \
    {                                                          \
        bitcompResult_t err = call;                            \
        if (BITCOMP_SUCCESS != err)                            \
        {                                                      \
            fprintf(                                           \
                stderr,                                        \
                "Bitcomp error %d in file '%s' in line %i.\n", \
                err,                                           \
                __FILE__,                                      \
                __LINE__);                                     \
            fflush(stderr);                                    \
            exit(EXIT_FAILURE);                                \
        }                                                      \
    }

float *compress(char *filePath)
{
    size_t fileSize = io::FileSize(filePath);
    float *inputHost = (float *)malloc(fileSize);
    io::read_binary_to_array<float>(filePath, inputHost, fileSize / sizeof(float));

    float *inputDevice;
    CUDA_CHECK(hipMalloc(&inputDevice, fileSize));
    CUDA_CHECK(hipMemcpy(inputDevice, inputHost, fileSize, hipMemcpyHostToDevice));

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Let's execute all the GPU code in a non-default stream
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    // Create a bitcomp plan to compress FP32 data using a signed integer
    // quantization, since the input data contains positive and negative values.
    bitcompHandle_t plan;
    BITCOMP_CHECK(bitcompCreatePlan(
        &plan,                  // Bitcomp handle
        fileSize,               // Size in bytes of the uncompressed data
        BITCOMP_FP32_DATA,      // Data type
        BITCOMP_LOSSLESS,       // Compression type
        BITCOMP_DEFAULT_ALGO)); // Bitcomp algo, default or sparse

    // Query the maximum size of the compressed data (worst case scenario)
    // and allocate the compressed buffer
    size_t maxlen = bitcompMaxBuflen(fileSize);
    void *compbuf;
    CUDA_CHECK(hipMalloc(&compbuf, maxlen));

    // Associate the bitcomp plan to the stream, otherwise the compression
    // or decompression would happen in the default stream
    BITCOMP_CHECK(bitcompSetStream(plan, stream));

    // Start recording on the specified stream
    hipEventRecord(start, stream);

    // Compress the input data with the chosen quantization delta
    BITCOMP_CHECK(bitcompCompressLossless(plan, inputDevice, compbuf));

    // Stop recording on the specified stream
    hipEventRecord(stop, stream);

    // Wait for the compression kernel to finish
    CUDA_CHECK(hipStreamSynchronize(stream));

    // Query the compressed size
    size_t compsize;
    BITCOMP_CHECK(bitcompGetCompressedSize(compbuf, &compsize));
    float ratio = static_cast<float>(fileSize) / static_cast<float>(compsize);
    printf("Compression ratio = %.2f\n", ratio);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Output the elapsed time
    std::cout << "Compression elapsed time: " << milliseconds << " ms" << std::endl;
    std::cout << "Compression throughput: " << static_cast<float>(fileSize) / 1024 / 1024 / milliseconds << " GB/s" << std::endl;

    char *compressedDataHost = (char *)malloc(compsize);
    CUDA_CHECK(hipMemcpy(compressedDataHost, compbuf, compsize, hipMemcpyDeviceToHost));
    std::string str(filePath);
    io::write_array_to_binary<char>(str + ".bitcomp", compressedDataHost, compsize);

    // Destroy the events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Clean up
    BITCOMP_CHECK(bitcompDestroyPlan(plan));
    CUDA_CHECK(hipFree(compbuf));
    CUDA_CHECK(hipFree(inputDevice));

    // free(inputHost);
    free(compressedDataHost);
    return inputHost;
}

float *decompress(char *filePath, size_t originalSize)
{
    size_t fileSize = io::FileSize(filePath);
    char *inputHost = (char *)malloc(fileSize);
    io::read_binary_to_array<char>(filePath, inputHost, fileSize / sizeof(char));

    char *inputDevice;
    CUDA_CHECK(hipMalloc(&inputDevice, fileSize));
    CUDA_CHECK(hipMemcpy(inputDevice, inputHost, fileSize, hipMemcpyHostToDevice));

    // Let's execute all the GPU code in a non-default stream
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate a buffer for the decompressed data
    float *outputDevice;
    CUDA_CHECK(hipMalloc(&outputDevice, originalSize));

    // Create a bitcomp plan to compress FP32 data using a signed integer
    // quantization, since the input data contains positive and negative values.
    bitcompHandle_t plan;
    BITCOMP_CHECK(bitcompCreatePlan(
        &plan,                  // Bitcomp handle
        originalSize,           // Size in bytes of the uncompressed data
        BITCOMP_FP32_DATA,      // Data type
        BITCOMP_LOSSLESS,       // Compression type
        BITCOMP_DEFAULT_ALGO)); // Bitcomp algo, default or sparse

    // Associate the bitcomp plan to the stream, otherwise the compression
    // or decompression would happen in the default stream
    BITCOMP_CHECK(bitcompSetStream(plan, stream));

    // Start recording on the specified stream
    hipEventRecord(start, stream);

    // Decompress the data
    BITCOMP_CHECK(bitcompUncompress(plan, inputDevice, outputDevice));

    // Stop recording on the specified stream
    hipEventRecord(stop, stream);

    // Wait for the decompression to finish
    CUDA_CHECK(hipStreamSynchronize(stream));

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Output the elapsed time
    std::cout << "Decompression elapsed time: " << milliseconds << " ms" << std::endl;
    std::cout << "Decompression throughput: " << static_cast<float>(originalSize) / 1024 / 1024 / milliseconds << " GB/s" << std::endl;

    float *outputHost = (float *)malloc(originalSize);
    CUDA_CHECK(hipMemcpy(outputHost, outputDevice, originalSize, hipMemcpyDeviceToHost));
    std::string str(filePath);
    io::write_array_to_binary<float>(str + ".decompressed", outputHost, originalSize / sizeof(float));

    // Clean up
    BITCOMP_CHECK(bitcompDestroyPlan(plan));
    CUDA_CHECK(hipFree(inputDevice));
    CUDA_CHECK(hipFree(outputDevice));

    free(inputHost);
    // free(outputHost);
    return outputHost;
}

void roundTripVerification(char *filePath)
{
    size_t fileSize = io::FileSize(filePath);
    float *originalData = compress(filePath);
    std::string str(filePath);
    str = str + ".bitcomp";
    char *compressedFilePath = (char *)str.c_str();
    float *reconstructedData = decompress(compressedFilePath, fileSize);

    for (int i = 0; i < fileSize / sizeof(float); i++)
    {
        if (originalData[i] != reconstructedData[i])
        {
            std::cout << "Error: originalData[" << i << "] = " << originalData[i] << " != reconstructedData[" << i << "] = " << reconstructedData[i] << std::endl;
            return;
        }
    }

    free(originalData);
    free(reconstructedData);
    return;
}

int main(int argc, char *argv[])
{
    if (strcmp(argv[1], "-c") == 0)
    {
        float *originalData = compress(argv[2]);
        free(originalData);
    }
    else if (strcmp(argv[1], "-d") == 0)
    {
        size_t originalSize = std::stoi(argv[3]);
        float *reconstructedData = decompress(argv[2], originalSize);
        free(reconstructedData);
    }
    else if (strcmp(argv[1], "-r") == 0)
    {
        roundTripVerification(argv[2]);
    }

    return 0;
}